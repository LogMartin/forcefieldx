#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <cutil_inline.h>
#include <jni.h>
#include "ffx_numerics_fft_Complex3DCuda.h"

__global__ void recipSummation(float* data, float* recip, int len)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) { 
       const int j = 2 * i;
       data[j]     *= recip[i];
       data[j + 1] *= recip[i];
    }
    __syncthreads();
}

/*
 * Class:     ffx_numerics_fft_Complex3DCuda
 * Method:    init
 * Signature: (III[F[F[J)I
 */
JNIEXPORT jint JNICALL Java_ffx_numerics_fft_Complex3DCuda_init
  (JNIEnv *env, jobject obj, jint nx, jint ny, jint nz, 
   jfloatArray dataArray, jfloatArray recipArray, jlongArray pointerArray) {

   // Init the CUDA device.
   hipSetDevice( cutGetMaxGflopsDeviceId() );

   // Compute the needed device memory.
   int len = nx * ny * nz;
   int dataSize = len * 2 * sizeof(float);
   int recipSize = len * sizeof(float);

   // Create the FFT plan and allocate device memory.
   float *d_data, *d_recip;
   hipfftHandle plan;
   cutilSafeCall(hipMalloc((void**)&d_data, dataSize));
   cutilSafeCall(hipMalloc((void**)&d_recip, recipSize));
   cufftSafeCall(hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_C2C));

   // Save the FFT plan ID and GPU memory addresses.
   jlong *pointers = (jlong*) env->GetPrimitiveArrayCritical(pointerArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   pointers[0] = plan;
   pointers[1] = (jlong) d_data;
   pointers[2] = (jlong) d_recip; 
   env->ReleasePrimitiveArrayCritical(pointerArray, pointers, 0);

   // Copy the data and reciprocal vectors to the GPU.
   jfloat *data = (jfloat*) env->GetPrimitiveArrayCritical(dataArray, 0);
   jfloat *recip = (jfloat*) env->GetPrimitiveArrayCritical(recipArray, 0);
   if (recip == NULL || data == NULL) {
       return -1;
   }
   cutilSafeCall(hipMemcpy(d_data, data, dataSize, hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(d_recip, recip, recipSize, hipMemcpyHostToDevice));
   env->ReleasePrimitiveArrayCritical(dataArray, data, 0);
   env->ReleasePrimitiveArrayCritical(recipArray, recip, 0);
   return 1;
}

/*
 * Class:     ffx_numerics_fft_Complex3DCuda
 * Method:    convolution
 * Signature: ([F[J)I
 */
JNIEXPORT jint JNICALL Java_ffx_numerics_fft_Complex3DCuda_convolution
  (JNIEnv *env, jobject obj, jfloatArray dataArray, jlongArray pointerArray) {

   // Get a reference to the data array
   jint len = env->GetArrayLength(dataArray) / 2; 
   jfloat *data = (jfloat*) env->GetPrimitiveArrayCritical(dataArray, 0);
   if (data == NULL) {
       return -1;
   }

   // Get the FFT plan ID and GPU memory addresses.
   jlong *pointers = (jlong*) env->GetPrimitiveArrayCritical(pointerArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   hipfftHandle plan = (hipfftHandle) pointers[0];
   float *d_data = (float*) pointers[1];
   float *d_recip = (float*) pointers[2];

   // Compute the needed GPU memory and numbers of thread blocks.
   int threads = 512;
   int blocks = (len + threads - 1) / threads;
   int dataSize = 2 * len * sizeof(float);

   // Copy the data to the GPU and do the convolution.
   cutilSafeCall(hipMemcpy(d_data, data, dataSize, hipMemcpyHostToDevice));
   cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *) d_data, (hipfftComplex *) d_data, HIPFFT_FORWARD));
   recipSummation<<<blocks,threads>>>(d_data, d_recip, len);
   cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *) d_data, (hipfftComplex *) d_data, HIPFFT_BACKWARD));
   cutilSafeCall(hipMemcpy(data, d_data, dataSize, hipMemcpyDeviceToHost));

   env->ReleasePrimitiveArrayCritical(pointerArray, pointers, 0);
   env->ReleasePrimitiveArrayCritical(dataArray, data, 0);
   return 1;
}

/*
 * Class:     ffx_numerics_fft_Complex3DCuda
 * Method:    free
 * Signature: ([J)I
 */
JNIEXPORT jint JNICALL Java_ffx_numerics_fft_Complex3DCuda_free
  (JNIEnv *env, jobject obj, jlongArray pointerArray) {

   jlong *pointers = (jlong*) env->GetPrimitiveArrayCritical(pointerArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   hipfftHandle plan = (hipfftHandle) pointers[0];
   float *d_data = (float*) pointers[1];
   float *d_recip = (float*) pointers[2];
   env->ReleasePrimitiveArrayCritical(pointerArray, pointers, 0);

   cufftSafeCall(hipfftDestroy(plan));
   cutilSafeCall(hipFree(d_data));
   cutilSafeCall(hipFree(d_recip));
   hipDeviceReset();
   return 1;
}

