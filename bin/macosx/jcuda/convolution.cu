#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <cutil_inline.h>
#include <jni.h>
#include "ffx_numerics_fft_Complex3DCuda.h"

static __global__ void recipSummation(float* data, float* recip, int len)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= len) return;
    const int j = 2 * i;
    data[j]     *= recip[i];
    data[j + 1] *= recip[i];
}

/*
 * Class:     ffx_numerics_fft_Complex3DCuda
 * Method:    init
 * Signature: (III[F[F[J)I
 */
JNIEXPORT jint JNICALL Java_ffx_numerics_fft_Complex3DCuda_init
  (JNIEnv *env, jobject obj, jint nx, jint ny, jint nz, 
   jfloatArray data, jfloatArray recipArray, jlongArray pointerArray) {

   // Init the CUDA device.
   hipSetDevice( cutGetMaxGflopsDeviceId() );

   // Compute the needed device memory.
   int len = nx * ny * nz;
   int dataSize = len * 2 * sizeof(float);
   int recipSize = len * sizeof(float);

   // Create the FFT plan and allocate device memory.
   float *d_data, *d_recip;
   hipfftHandle plan;
   cutilSafeCall(hipMalloc((void**)&d_data, dataSize));
   cutilSafeCall(hipMalloc((void**)&d_recip, recipSize));
   cufftSafeCall(hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_C2C));

   // Save the FFT plan ID and GPU memory addresses.
   jlong *pointers = (jlong*) env->GetPrimitiveArrayCritical(pointerArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   pointers[0] = plan;
   pointers[1] = (jlong) d_data;
   pointers[2] = (jlong) d_recip; 
   env->ReleasePrimitiveArrayCritical(pointerArray, pointers, 0);

   // Copy the reciprocal vector to the GPU.
   jfloat *recip = (jfloat*) env->GetPrimitiveArrayCritical(recipArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   cutilSafeCall(hipMemcpy(d_recip, recip, recipSize, hipMemcpyHostToDevice));
   env->ReleasePrimitiveArrayCritical(recipArray, recip, 0);
   return 1;
}

/*
 * Class:     ffx_numerics_fft_Complex3DCuda
 * Method:    convolution
 * Signature: ([F[J)I
 */
JNIEXPORT jint JNICALL Java_ffx_numerics_fft_Complex3DCuda_convolution
  (JNIEnv *env, jobject obj, jfloatArray dataArray, jlongArray pointerArray) {

   // Get the FFT plan ID and GPU memory addresses.
   jlong *pointers = (jlong*) env->GetPrimitiveArrayCritical(pointerArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   hipfftHandle plan = (hipfftHandle) pointers[0];
   float *d_data = (float*) pointers[1];
   float *d_recip = (float*) pointers[2];
   env->ReleasePrimitiveArrayCritical(pointerArray, pointers, 0);

   // Compute the needed GPU memory and numbers of thread blocks.
   jint len = env->GetArrayLength(dataArray) / 2; 
   int threads = 256;
   int blocks = (len + threads - 1) / threads;
   int dataSize = 2 * len * sizeof(float);

   // Copy the data to the GPU and do the convolution.
   jfloat *data = (jfloat*) env->GetPrimitiveArrayCritical(dataArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   cutilSafeCall(hipMemcpy(d_data, data, dataSize, hipMemcpyHostToDevice));
   cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *) d_data, (hipfftComplex *) d_data, HIPFFT_FORWARD));
   recipSummation<<<blocks,threads>>>(d_data, d_recip, len);
   cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *) d_data, (hipfftComplex *) d_data, HIPFFT_BACKWARD));
   cutilSafeCall(hipMemcpy(data, d_data, dataSize, hipMemcpyDeviceToHost));
   env->ReleasePrimitiveArrayCritical(dataArray, data, 0);
   return 1;
}

/*
 * Class:     ffx_numerics_fft_Complex3DCuda
 * Method:    free
 * Signature: ([J)I
 */
JNIEXPORT jint JNICALL Java_ffx_numerics_fft_Complex3DCuda_free
  (JNIEnv *env, jobject obj, jlongArray pointerArray) {

   jlong *pointers = (jlong*) env->GetPrimitiveArrayCritical(pointerArray, 0);
   if (pointers == NULL) {
       return -1;
   }
   hipfftHandle plan = (hipfftHandle) pointers[0];
   float *d_data = (float*) pointers[1];
   float *d_recip = (float*) pointers[2];
   env->ReleasePrimitiveArrayCritical(pointerArray, pointers, 0);

   cufftSafeCall(hipfftDestroy(plan));
   cutilSafeCall(hipFree(d_data));
   cutilSafeCall(hipFree(d_recip));
   hipDeviceReset();
   return 1;
}

