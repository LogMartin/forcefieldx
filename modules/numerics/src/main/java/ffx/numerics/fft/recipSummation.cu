
#include <hip/hip_runtime.h>
extern "C"
__global__ void recipSummation(float* data, float* recip, int len)
{
    const int y = blockIdx.y * gridDim.x * blockDim.x;
    const int x = blockIdx.x * blockDim.x;
    const int i = threadIdx.x + x + y;
    if (i < len) {
       const int j = 2 * i;
       data[j]     *= recip[i];
       data[j + 1] *= recip[i];
    }
}
