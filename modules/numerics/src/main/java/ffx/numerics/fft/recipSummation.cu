
#include <hip/hip_runtime.h>
extern "C"
__global__ void recipSummation(float* data, float* recip, int len)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
       const int j = 2 * i;
       data[j]     *= recip[i];
       data[j + 1] *= recip[i];
    }
}
